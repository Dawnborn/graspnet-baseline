#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: points(b, c, n)(1 3 20000)
// idx: (b, npoints, nsample)(1 1024 64)
// output: out(b, c, npoints, nsample)(1 3 1024 64)
// block size: (1024, 3)
// 已知点在原始点云中的idx，将对应的点填入out
__global__ void group_points_kernel(int b, int c, int n, int npoints,
                                    int nsample,
                                    const float *__restrict__ points,
                                    const int *__restrict__ idx,
                                    float *__restrict__ out) {
  int batch_index = blockIdx.x;
  points += batch_index * n * c;
  idx += batch_index * npoints * nsample;
  out += batch_index * npoints * nsample * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      out[(l * npoints + j) * nsample + k] = points[l * n + ii];
    }
  }
}

void group_points_kernel_wrapper(int b, int c, int n, int npoints,
                                 int nsample,  // 1 3 20000 1024 64
                                 const float *points, const int *idx,
                                 float *out) {  // 1024*64
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // block 尺寸为1024，3 (num_seed, 3)
  group_points_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
      b, c, n, npoints, nsample, points, idx, out);

  CUDA_CHECK_ERRORS();
}

// input: grad_out(b, c, npoints, nsample), idx(b, npoints, nsample)
// output: grad_points(b, c, n)
__global__ void group_points_grad_kernel(int b, int c, int n, int npoints,
                                         int nsample,
                                         const float *__restrict__ grad_out,
                                         const int *__restrict__ idx,
                                         float *__restrict__ grad_points) {
  int batch_index = blockIdx.x;
  grad_out += batch_index * npoints * nsample * c;
  idx += batch_index * npoints * nsample;
  grad_points += batch_index * n * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      atomicAdd(grad_points + l * n + ii,
                grad_out[(l * npoints + j) * nsample + k]);
    }
  }
}

void group_points_grad_kernel_wrapper(int b, int c, int n, int npoints,
                                      int nsample, const float *grad_out,
                                      const int *idx, float *grad_points) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  group_points_grad_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
      b, c, n, npoints, nsample, grad_out, idx, grad_points);

  CUDA_CHECK_ERRORS();
}
